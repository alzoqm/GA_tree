#include "hip/hip_runtime.h"
// --- START OF FILE csrc/predict_kernel.cu ---

// csrc/predict_kernel.cu
#include <hip/hip_runtime.h>
#include "constants.h"

// --- Device-level Helper Function --- (수정 없음)
__device__ bool evaluate_node_device(
    const float* node_data,
    const float* feature_values) {

    int comp_type = static_cast<int>(node_data[COL_PARAM_3]);
    int feat1_idx = static_cast<int>(node_data[COL_PARAM_1]);
    float val1 = feature_values[feat1_idx];
    float val2 = node_data[COL_PARAM_4];

    if (comp_type == COMP_TYPE_FEAT_FEAT) {
        int feat2_idx = static_cast<int>(node_data[COL_PARAM_4]);
        val2 = feature_values[feat2_idx];
    }
    
    if (comp_type == COMP_TYPE_FEAT_BOOL) {
        return val1 == val2;
    }

    int op = static_cast<int>(node_data[COL_PARAM_2]);
    switch(op) {
        case OP_GTE: return val1 >= val2;
        case OP_LTE: return val1 <= val2;
    }

    return false;
}


// --- Main CUDA Kernel ---
__global__ void predict_kernel(
    const float* population_ptr,
    const float* features_ptr, // 이 포인터는 이제 (num_features) 크기의 1D 배열을 가리킵니다.
    const long* positions_ptr,
    const int* next_indices_ptr,
    float* results_ptr,
    int pop_size,
    int max_nodes,
    int num_features) {

    // 1. Thread-to-Tree Mapping
    const int tree_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tree_idx >= pop_size) {
        return;
    }

    // 2. Setup pointers and variables for the current tree
    const float* tree_data = population_ptr + tree_idx * max_nodes * NODE_INFO_DIM;
    
    // [수정] features 포인터를 인덱싱하지 않고 그대로 사용합니다.
    // 모든 스레드는 동일한 features 포인터를 공유합니다.
    const float* features = features_ptr;
    
    float* result_out = results_ptr + tree_idx * 4;
    const int next_idx = next_indices_ptr[tree_idx];

    // 3. Find the starting node (root branch)
    long start_pos_type = positions_ptr[tree_idx];
    int start_node_idx = -1;
    for (int i = 0; i < 3; ++i) { 
        const float* node = tree_data + i * NODE_INFO_DIM;
        if (static_cast<int>(node[COL_NODE_TYPE]) == NODE_TYPE_ROOT_BRANCH &&
            static_cast<int>(node[COL_PARAM_1]) == start_pos_type) {
            start_node_idx = i;
            break;
        }
    }

    // 기본 결과값(HOLD) 설정
    result_out[0] = ACTION_NOT_FOUND;
    result_out[1] = 0.0f;
    result_out[2] = 0.0f;
    result_out[3] = 0.0f;

    if (start_node_idx == -1) {
        return;
    }

    // 4. BFS(너비 우선 탐색)를 위한 로컬 원형 큐
    int bfs_queue[2048];
    int queue_head = 0;
    int queue_tail = 0;

    if (queue_tail < 2048) {
        bfs_queue[queue_tail++] = start_node_idx;
    }
    
    bool found_action = false;

    // 5. BFS 루프 시작 (수정 없음)
    while (queue_head < queue_tail && !found_action) {
        int current_node_idx = bfs_queue[queue_head++];

        for (int child_idx = 0; child_idx < next_idx; ++child_idx) {
            const float* child_node_data = tree_data + child_idx * NODE_INFO_DIM;

            if (static_cast<int>(child_node_data[COL_PARENT_IDX]) == current_node_idx) {
                int child_node_type = static_cast<int>(child_node_data[COL_NODE_TYPE]);

                if (child_node_type == NODE_TYPE_ACTION) {
                    result_out[0] = child_node_data[COL_PARAM_1];
                    result_out[1] = child_node_data[COL_PARAM_2];
                    result_out[2] = child_node_data[COL_PARAM_3];
                    result_out[3] = child_node_data[COL_PARAM_4];
                    found_action = true;
                    break;
                }
                
                else if (child_node_type == NODE_TYPE_DECISION) {
                    if (evaluate_node_device(child_node_data, features)) {
                        if (queue_tail < 2048) {
                            bfs_queue[queue_tail++] = child_idx;
                        }
                    }
                }
            }
        }
    }
}

// --- Kernel Launcher --- (수정 없음)
void launch_predict_kernel(
    const float* population_ptr,
    const float* features_ptr,
    const long* positions_ptr,
    const int* next_indices_ptr,
    float* results_ptr,
    int pop_size,
    int max_nodes,
    int num_features) {

    if (pop_size == 0) return;

    const int threads_per_block = 256;
    const int num_blocks = (pop_size + threads_per_block - 1) / threads_per_block;

    predict_kernel<<<num_blocks, threads_per_block>>>(
        population_ptr,
        features_ptr,
        positions_ptr,
        next_indices_ptr,
        results_ptr,
        pop_size,
        max_nodes,
        num_features
    );
}

// --- END OF FILE csrc/predict_kernel.cu ---