#include "hip/hip_runtime.h"
// csrc/predict_kernel.cu (수정됨)
#include <hip/hip_runtime.h>
#include "constants.h"

// [신규] 공유 메모리에 캐싱할 피처의 최대 개수.
// 실제 피처 수가 이보다 많으면 에러가 발생합니다. (C++ 래퍼에서 체크)
constexpr int MAX_FEATURES_IN_SHARED_MEM = 1024;

// --- Device-level Helper Function ---
__device__ bool evaluate_node_device(
    const float* node_data,
    const float* feature_values) { // 이제 feature_values는 공유 메모리 포인터가 됩니다.

    int comp_type = static_cast<int>(node_data[COL_PARAM_3]);
    int feat1_idx = static_cast<int>(node_data[COL_PARAM_1]);
    float val1 = feature_values[feat1_idx];
    float val2 = node_data[COL_PARAM_4];

    if (comp_type == COMP_TYPE_FEAT_FEAT) {
        int feat2_idx = static_cast<int>(node_data[COL_PARAM_4]);
        val2 = feature_values[feat2_idx];
    }
    
    if (comp_type == COMP_TYPE_FEAT_BOOL) {
        return val1 == val2;
    }

    int op = static_cast<int>(node_data[COL_PARAM_2]);
    switch(op) {
        case OP_GTE: return val1 >= val2;
        case OP_LTE: return val1 <= val2;
    }

    return false;
}


// --- [수정] Main CUDA Kernel ---
__global__ void predict_kernel(
    const float* population_ptr,
    const float* features_ptr,
    const long* positions_ptr,
    const int* next_indices_ptr,
    // [신규] 인접 리스트 포인터
    const int* offset_ptr,
    const int* child_indices_ptr,
    float* results_ptr,
    int* bfs_queue_buffer,
    int pop_size,
    int max_nodes,
    int num_features) {

    // [신규] 공유 메모리 선언
    __shared__ float feature_cache[MAX_FEATURES_IN_SHARED_MEM];

    // 1. Thread-to-Tree Mapping
    const int tree_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tree_idx >= pop_size) {
        return;
    }
    
    // [신규] 블록의 첫 번째 스레드가 피처 값을 공유 메모리로 캐싱
    if (threadIdx.x < num_features) {
        feature_cache[threadIdx.x] = features_ptr[threadIdx.x];
    }
    __syncthreads(); // 블록 내 모든 스레드가 캐싱 완료까지 대기

    // 2. Setup pointers and variables for the current tree
    const float* tree_data = population_ptr + tree_idx * max_nodes * NODE_INFO_DIM;
    float* result_out = results_ptr + tree_idx * 4;
    const int next_idx = next_indices_ptr[tree_idx];
    // [신규] 현재 트리에 해당하는 오프셋 배열 부분 포인터 설정
    const int* tree_offset_ptr = offset_ptr + tree_idx * max_nodes;

    // 3. Find the starting node (root branch)
    long start_pos_type = positions_ptr[tree_idx];
    int start_node_idx = -1;
    for (int i = 0; i < 3; ++i) { 
        const float* node = tree_data + i * NODE_INFO_DIM;
        if (static_cast<int>(node[COL_NODE_TYPE]) == NODE_TYPE_ROOT_BRANCH &&
            static_cast<int>(node[COL_PARAM_1]) == start_pos_type) {
            start_node_idx = i;
            break;
        }
    }

    result_out[0] = ACTION_NOT_FOUND;
    result_out[1] = 0.0f;
    result_out[2] = 0.0f;
    result_out[3] = 0.0f;

    if (start_node_idx == -1) {
        return;
    }

    // 4. BFS를 위한 큐 관리 변수 설정
    int* bfs_queue = bfs_queue_buffer + tree_idx * max_nodes;
    int queue_head = 0;
    int queue_tail = 0;

    if (queue_tail < max_nodes) {
        bfs_queue[queue_tail++] = start_node_idx;
    }
    
    bool found_action = false;

    // 5. BFS 루프 시작
    while (queue_head < queue_tail && !found_action) {
        int current_node_idx = bfs_queue[queue_head++];

        // [수정] 비효율적인 선형 탐색을 인접 리스트 조회로 변경 (핵심 최적화)
        int start_offset = tree_offset_ptr[current_node_idx];
        int end_offset = tree_offset_ptr[current_node_idx + 1];

        for (int i = start_offset; i < end_offset; ++i) {
            int child_idx = child_indices_ptr[i];
            const float* child_node_data = tree_data + child_idx * NODE_INFO_DIM;
            int child_node_type = static_cast<int>(child_node_data[COL_NODE_TYPE]);

            if (child_node_type == NODE_TYPE_ACTION) {
                result_out[0] = child_node_data[COL_PARAM_1];
                result_out[1] = child_node_data[COL_PARAM_2];
                result_out[2] = child_node_data[COL_PARAM_3];
                result_out[3] = child_node_data[COL_PARAM_4];
                found_action = true;
                break; // Action을 찾았으므로 더 이상 자식을 볼 필요 없음
            }
            else if (child_node_type == NODE_TYPE_DECISION) {
                // [수정] 공유 메모리에 캐시된 피처 값을 사용
                if (evaluate_node_device(child_node_data, feature_cache)) {
                    if (queue_tail < max_nodes) {
                        bfs_queue[queue_tail++] = child_idx;
                    }
                }
            }
        }
    }
}

// --- [수정] Kernel Launcher ---
void launch_predict_kernel(
    const float* population_ptr,
    const float* features_ptr,
    const long* positions_ptr,
    const int* next_indices_ptr,
    // [신규] 인접 리스트 포인터
    const int* offset_ptr,
    const int* child_indices_ptr,
    float* results_ptr,
    int* bfs_queue_buffer_ptr,
    int pop_size,
    int max_nodes,
    int num_features) {

    if (pop_size == 0) return;

    const int threads_per_block = 256;
    const int num_blocks = (pop_size + threads_per_block - 1) / threads_per_block;
    
    predict_kernel<<<num_blocks, threads_per_block>>>(
        population_ptr,
        features_ptr,
        positions_ptr,
        next_indices_ptr,
        offset_ptr,
        child_indices_ptr,
        results_ptr,
        bfs_queue_buffer_ptr,
        pop_size,
        max_nodes,
        num_features
    );
}