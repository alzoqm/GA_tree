#include "hip/hip_runtime.h"
// csrc/crossover_kernel.cu
#include "crossover_kernel.cuh"
#include "constants.h"
#include <hiprand/hiprand_kernel.h>

// ==============================================================================
//           커널 1: 컨텍스트 마스크 생성 (Contextual Mask Generation)
// ==============================================================================
__global__ void get_contextual_mask_kernel(
    const float* trees_ptr,
    bool* output_mask_ptr,
    int batch_size,
    int max_nodes,
    int node_type_target,
    int branch_type_target)
{
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_nodes = batch_size * max_nodes;
    if (gid >= total_nodes) return;

    const int batch_idx = gid / max_nodes;
    const int node_idx = gid % max_nodes;

    const float* node_data = trees_ptr + gid * NODE_INFO_DIM;
    
    // 1. 노드 타입이 일치하는지 확인
    if ((int)node_data[COL_NODE_TYPE] == node_type_target) {
        
        // 2. 루트까지 부모를 거슬러 올라가기
        int current_idx = node_idx;
        int parent_idx = (int)node_data[COL_PARENT_IDX];

        while (parent_idx != -1) {
            current_idx = parent_idx;
            parent_idx = (int)trees_ptr[(batch_idx * max_nodes + current_idx) * NODE_INFO_DIM + COL_PARENT_IDX];
        }

        // 3. 루트 분기 타입이 일치하는지 확인
        const float* root_data = trees_ptr + (batch_idx * max_nodes + current_idx) * NODE_INFO_DIM;
        if ((int)root_data[COL_PARAM_1] == branch_type_target) {
            output_mask_ptr[gid] = true;
        }
    }
}

// ==============================================================================
//         커널 2: 노드 파라미터 교환 (Node Parameter Swap)
// ==============================================================================
__global__ void swap_node_params_kernel(
    float* c1_ptr,
    float* c2_ptr,
    const bool* p1_mask_ptr,
    const bool* p2_mask_ptr,
    int batch_size,
    int max_nodes)
{
    const int batch_idx = blockIdx.x;
    if (batch_idx >= batch_size) return;

    // --- 1. 교환 가능한 노드 수 계산 ---
    int p1_count = 0;
    int p2_count = 0;
    for (int i = 0; i < max_nodes; ++i) {
        if (p1_mask_ptr[batch_idx * max_nodes + i]) p1_count++;
        if (p2_mask_ptr[batch_idx * max_nodes + i]) p2_count++;
    }

    int max_swaps = min(p1_count, p2_count);
    if (max_swaps == 0) return;

    // --- 2. 교환할 개수 'k' 결정 ---
    hiprandState state;
    hiprand_init(batch_idx, 0, 0, &state);
    int k_upper = max(1, max_swaps / 2);
    int k = (int)(hiprand_uniform(&state) * k_upper) + 1;
    k = min(k, max_swaps);

    // --- 3. 교환 수행 (단일 스레드 내 루프) ---
    for (int i = 0; i < k; ++i) {
        // p1에서 랜덤 인덱스 선택
        int p1_rand_n = (int)(hiprand_uniform(&state) * p1_count);
        int p1_swap_idx = -1;
        int current_n = 0;
        for (int j = 0; j < max_nodes; ++j) {
            if (p1_mask_ptr[batch_idx * max_nodes + j]) {
                if (current_n == p1_rand_n) {
                    p1_swap_idx = j;
                    break;
                }
                current_n++;
            }
        }

        // p2에서 랜덤 인덱스 선택
        int p2_rand_n = (int)(hiprand_uniform(&state) * p2_count);
        int p2_swap_idx = -1;
        current_n = 0;
        for (int j = 0; j < max_nodes; ++j) {
            if (p2_mask_ptr[batch_idx * max_nodes + j]) {
                if (current_n == p2_rand_n) {
                    p2_swap_idx = j;
                    break;
                }
                current_n++;
            }
        }

        if (p1_swap_idx != -1 && p2_swap_idx != -1) {
            // 파라미터(COL_PARAM_1 ~ 끝) 교환
            for (int param_col = COL_PARAM_1; param_col < NODE_INFO_DIM; ++param_col) {
                float* p1_addr = c1_ptr + (batch_idx * max_nodes + p1_swap_idx) * NODE_INFO_DIM + param_col;
                float* p2_addr = c2_ptr + (batch_idx * max_nodes + p2_swap_idx) * NODE_INFO_DIM + param_col;
                
                float temp = *p1_addr;
                *p1_addr = *p2_addr;
                *p2_addr = temp;
            }
        }
    }
}

__device__ int find_subtree_nodes_device(
    const float* tree_ptr,
    int root_idx,
    int* queue_buffer,      // 스레드별 BFS 큐
    int* result_indices,    // 스레드별 결과 저장 버퍼
    int max_nodes)
{
    if (root_idx < 0 || root_idx >= max_nodes || tree_ptr[root_idx * NODE_INFO_DIM + COL_NODE_TYPE] == NODE_TYPE_UNUSED) {
        return 0;
    }
    
    int head = 0, tail = 0;
    queue_buffer[tail++] = root_idx;
    result_indices[0] = root_idx;
    int count = 1;

    while (head < tail) {
        int current_idx = queue_buffer[head++];
        
        for (int i = 0; i < max_nodes; ++i) {
            if ((int)tree_ptr[i * NODE_INFO_DIM + COL_PARENT_IDX] == current_idx) {
                if (tail < max_nodes) { // 버퍼 오버플로우 방지
                    queue_buffer[tail++] = i;
                    if (count < max_nodes) { // 버퍼 오버플로우 방지
                        result_indices[count++] = i;
                    }
                } else {
                    return count;
                }
            }
        }
    }
    return count;
}


// ==============================================================================
//       [신규] 커널 3: RootBranchCrossover를 위한 배치 복사 커널
// ==============================================================================
__global__ void copy_branches_kernel(
    float* child_batch_ptr,
    const float* p1_batch_ptr,
    const float* p2_batch_ptr,
    const int* donor_map_ptr,
    int* bfs_queue_buffer_ptr,
    int* result_indices_buffer_ptr,
    int* old_to_new_map_buffer_ptr,
    int batch_size,
    int max_nodes)
{
    const int batch_idx = blockIdx.x;
    if (batch_idx >= batch_size) return;

    // [안전성] child 버퍼 중 non-root 노드만 UNUSED로 초기화하여 잔존 쓰레기 노드 방지
    // 루트 브랜치(0,1,2)는 Python에서 이미 설정되었으므로 건드리지 않음
    float* child_ptr_init = child_batch_ptr + batch_idx * max_nodes * NODE_INFO_DIM;
    for (int i = 3; i < max_nodes; ++i) {  // Start from index 3, skip root branches (0,1,2)
        float* nd = child_ptr_init + i * NODE_INFO_DIM;
        nd[COL_NODE_TYPE] = NODE_TYPE_UNUSED;
        // 부모/깊이/파라미터는 필요 시 이후 복사에서 덮어씀
    }

    // --- 1. 스레드별 포인터 및 버퍼 설정 ---
    float* child_ptr = child_batch_ptr + batch_idx * max_nodes * NODE_INFO_DIM;
    const float* p1_ptr = p1_batch_ptr + batch_idx * max_nodes * NODE_INFO_DIM;
    const float* p2_ptr = p2_batch_ptr + batch_idx * max_nodes * NODE_INFO_DIM;
    
    int* my_queue = bfs_queue_buffer_ptr + batch_idx * max_nodes;
    int* my_results = result_indices_buffer_ptr + batch_idx * max_nodes;
    int* my_old_to_new_map = old_to_new_map_buffer_ptr + batch_idx * max_nodes;

    int child_next_idx = 3; 

    // --- 2. 3개의 브랜치(LONG, HOLD, SHORT)에 대해 순차적으로 복사 ---
    for (int b_idx = 0; b_idx < 3; ++b_idx) {
        int donor_choice = donor_map_ptr[batch_idx * 3 + b_idx];
        const float* donor_ptr = (donor_choice == 0) ? p1_ptr : p2_ptr;

        for (int donor_node_idx = 0; donor_node_idx < max_nodes; ++donor_node_idx) {
            if ((int)donor_ptr[donor_node_idx * NODE_INFO_DIM + COL_PARENT_IDX] == b_idx) {
                int subtree_root_idx = donor_node_idx;
                
                int subtree_size = find_subtree_nodes_device(donor_ptr, subtree_root_idx, my_queue, my_results, max_nodes);

                if (child_next_idx + subtree_size > max_nodes) {
                    continue; 
                }

                for(int i=0; i<max_nodes; ++i) my_old_to_new_map[i] = -1;
                for (int k = 0; k < subtree_size; ++k) {
                    my_old_to_new_map[my_results[k]] = child_next_idx + k;
                }

                float dest_parent_depth = child_ptr[b_idx * NODE_INFO_DIM + COL_DEPTH];
                float source_root_depth = donor_ptr[subtree_root_idx * NODE_INFO_DIM + COL_DEPTH];
                float depth_offset = (dest_parent_depth + 1) - source_root_depth;

                for (int k = 0; k < subtree_size; ++k) {
                    int old_idx = my_results[k];
                    int new_idx = my_old_to_new_map[old_idx];
                    
                    const float* src_node_data = donor_ptr + old_idx * NODE_INFO_DIM;
                    float* dest_node_data = child_ptr + new_idx * NODE_INFO_DIM;

                    for(int d=0; d<NODE_INFO_DIM; ++d) dest_node_data[d] = src_node_data[d];

                    dest_node_data[COL_DEPTH] += depth_offset;
                    
                    int old_parent_idx = (int)src_node_data[COL_PARENT_IDX];
                    if (old_idx == subtree_root_idx) {
                        dest_node_data[COL_PARENT_IDX] = (float)b_idx;
                    } else {
                        dest_node_data[COL_PARENT_IDX] = (float)my_old_to_new_map[old_parent_idx];
                    }
                }
                child_next_idx += subtree_size;
            }
        }
    }
}

// ==============================================================================
//       [수정된] 커널 4: SubtreeCrossover를 위한 배치 커널 및 헬퍼 함수
// ==============================================================================

__device__ int get_root_branch_type_device(const float* tree_ptr, int node_idx, int max_nodes) {
    int current_idx = node_idx;
    while (current_idx >= 0 && current_idx < max_nodes) {
        int parent_idx = (int)tree_ptr[current_idx * NODE_INFO_DIM + COL_PARENT_IDX];
        if (parent_idx == -1) {
            return (int)tree_ptr[current_idx * NODE_INFO_DIM + COL_PARAM_1];
        }
        current_idx = parent_idx;
    }
    return -1; // Error or root not found
}

__device__ int get_active_node_count(const float* tree_ptr, int max_nodes) {
    int count = 0;
    for (int i = 0; i < max_nodes; ++i) {
        if (tree_ptr[i * NODE_INFO_DIM + COL_NODE_TYPE] != NODE_TYPE_UNUSED) {
            count++;
        }
    }
    return count;
}

__device__ int get_max_relative_depth(const float* tree_ptr, int* indices, int count, int max_nodes) {
    if (count == 0) return 0;
    int root_node_idx = indices[0];
    if (root_node_idx < 0 || root_node_idx >= max_nodes) return 0;

    float root_depth = tree_ptr[root_node_idx * NODE_INFO_DIM + COL_DEPTH];
    float max_abs_depth = root_depth;
    for (int i = 1; i < count; ++i) {
        int current_node_idx = indices[i];
        if (current_node_idx < 0 || current_node_idx >= max_nodes) continue;
        float current_depth = tree_ptr[current_node_idx * NODE_INFO_DIM + COL_DEPTH];
        if (current_depth > max_abs_depth) {
            max_abs_depth = current_depth;
        }
    }
    return (int)(max_abs_depth - root_depth);
}

__device__ bool would_violate_tree_structure(
    const float* child_ptr, int parent_idx, int new_subtree_root_type, int max_nodes)
{
    if (parent_idx < 0 || parent_idx >= max_nodes) return true;
    
    // Count existing children by type
    int action_children = 0;
    int decision_children = 0;
    
    for (int i = 0; i < max_nodes; ++i) {
        if ((int)child_ptr[i * NODE_INFO_DIM + COL_PARENT_IDX] == parent_idx) {
            int child_type = (int)child_ptr[i * NODE_INFO_DIM + COL_NODE_TYPE];
            if (child_type == NODE_TYPE_ACTION) {
                action_children++;
            } else if (child_type == NODE_TYPE_DECISION) {
                decision_children++;
            }
        }
    }
    
    // Check if adding new subtree would violate rules
    if (new_subtree_root_type == NODE_TYPE_ACTION) {
        // Rule 1: No mixed children (action + decision)
        if (decision_children > 0) return true;
        
        // Rule 2: Parent with action child must have exactly one child
        if (action_children > 0) return true; // Already has action child, can't add another
    } else if (new_subtree_root_type == NODE_TYPE_DECISION) {
        // Rule 1: No mixed children (action + decision)  
        if (action_children > 0) return true;
    }
    
    return false;
}

__device__ bool transplant_one_way_device(
    float* child_ptr, const float* recipient_ptr, const float* donor_ptr,
    int r_idx, int d_idx, int* r_indices, int r_count, int* d_indices, int d_count,
    int* my_old_to_new_map, int* my_empty_slots_buffer, int max_nodes)
{
    // 1. recipient로부터 child를 초기화
    for(int i=0; i < max_nodes * NODE_INFO_DIM; ++i) child_ptr[i] = recipient_ptr[i];

    // 2. child에서 제거될 서브트리 영역을 비움
    for (int i = 0; i < r_count; ++i) {
        int idx_to_clear = r_indices[i];
        if (idx_to_clear >= 0 && idx_to_clear < max_nodes) {
            child_ptr[idx_to_clear * NODE_INFO_DIM + COL_NODE_TYPE] = NODE_TYPE_UNUSED;
        }
    }

    // 3. 비어있는 슬롯 찾기
    int empty_count = 0;
    for (int i = 0; i < max_nodes && empty_count < d_count; ++i) {
        if (child_ptr[i * NODE_INFO_DIM + COL_NODE_TYPE] == NODE_TYPE_UNUSED) {
            my_empty_slots_buffer[empty_count++] = i;
        }
    }
    if (empty_count < d_count) return false; // 공간 부족 시 이식 중단

    // 4. old_to_new_map 생성
    for (int i = 0; i < max_nodes; ++i) my_old_to_new_map[i] = -1;
    for (int i = 0; i < d_count; ++i) {
        my_old_to_new_map[d_indices[i]] = my_empty_slots_buffer[i];
    }
    
    // 5. 깊이 오프셋 계산
    int r_parent_idx = (int)recipient_ptr[r_idx * NODE_INFO_DIM + COL_PARENT_IDX];
    if(r_parent_idx < 0 || r_parent_idx >= max_nodes) return false; // 유효하지 않은 부모
    float insertion_depth = recipient_ptr[r_parent_idx * NODE_INFO_DIM + COL_DEPTH] + 1.0f;
    float depth_offset = insertion_depth - donor_ptr[d_idx * NODE_INFO_DIM + COL_DEPTH];

    // 5.5. Check if transplantation would violate tree structure rules
    int donor_root_type = (int)donor_ptr[d_idx * NODE_INFO_DIM + COL_NODE_TYPE];
    if (would_violate_tree_structure(child_ptr, r_parent_idx, donor_root_type, max_nodes)) {
        return false; // Abort transplantation to avoid invalid structure
    }

    // 6. 노드 복사 및 업데이트
    for (int i = 0; i < d_count; ++i) {
        int old_idx = d_indices[i];
        int new_idx = my_old_to_new_map[old_idx];
        if (new_idx == -1) continue;

        const float* src_node = donor_ptr + old_idx * NODE_INFO_DIM;
        float* dest_node = child_ptr + new_idx * NODE_INFO_DIM;

        for(int d=0; d<NODE_INFO_DIM; ++d) dest_node[d] = src_node[d];
        dest_node[COL_DEPTH] += depth_offset;
        
        int old_parent_idx = (int)src_node[COL_PARENT_IDX];
        if (old_idx == d_idx) {
            dest_node[COL_PARENT_IDX] = (float)r_parent_idx;
        } else {
            dest_node[COL_PARENT_IDX] = (float)my_old_to_new_map[old_parent_idx];
        }
    }
    return true; // Success
}


__global__ void subtree_crossover_kernel(
    float* child1_out_ptr, float* child2_out_ptr,
    const float* p1_batch_ptr, const float* p2_batch_ptr,
    int mode, int max_depth, int max_nodes, int max_retries,
    const int* branch_perm_ptr,
    int* bfs_queue_buffer_ptr, int* result_indices_buffer_ptr, int* old_to_new_map_buffer_ptr,
    int* p1_candidates_buffer_ptr, int* p2_candidates_buffer_ptr,
    int batch_size)
{
    const int batch_idx = blockIdx.x;
    if (batch_idx >= batch_size) return;

    // --- 1. 스레드별 포인터 및 버퍼 설정 ---
    float* c1_ptr = child1_out_ptr + batch_idx * max_nodes * NODE_INFO_DIM;
    float* c2_ptr = child2_out_ptr + batch_idx * max_nodes * NODE_INFO_DIM;
    const float* p1_ptr = p1_batch_ptr + batch_idx * max_nodes * NODE_INFO_DIM;
    const float* p2_ptr = p2_batch_ptr + batch_idx * max_nodes * NODE_INFO_DIM;

    // [중요 수정] p1, p2 서브트리를 동시에 담기 위해 버퍼를 2*max_nodes로 사용
    //   - 래퍼에서 bfs_queue_buffer/result_indices_buffer는 [batch, 2*max_nodes]로 할당되어야 함
    int* my_queue1   = bfs_queue_buffer_ptr     + batch_idx * (2 * max_nodes);
    int* my_queue2   = my_queue1                + max_nodes;
    int* my_results1 = result_indices_buffer_ptr+ batch_idx * (2 * max_nodes);
    int* my_results2 = my_results1              + max_nodes;

    int* my_old_to_new_map = old_to_new_map_buffer_ptr + batch_idx * max_nodes;
    int* p1_candidates = p1_candidates_buffer_ptr + batch_idx * max_nodes;
    int* p2_candidates = p2_candidates_buffer_ptr + batch_idx * max_nodes;

    hiprandState state;
    hiprand_init(batch_idx, 0, 0, &state);

    bool success = false;
    for (int retry = 0; retry < max_retries && !success; ++retry) {
        // --- 2. 교차 후보군 선택 ---
        int p1_cand_count = 0, p2_cand_count = 0;
        
        if (mode == 0) { // free mode
            for(int i=0; i<max_nodes; ++i) {
                if(p1_ptr[i*NODE_INFO_DIM + COL_PARENT_IDX] != -1) p1_candidates[p1_cand_count++] = i;
                if(p2_ptr[i*NODE_INFO_DIM + COL_PARENT_IDX] != -1) p2_candidates[p2_cand_count++] = i;
            }
        } else { // context mode
            int branch_type_to_try = branch_perm_ptr[batch_idx * 3 + (retry % 3)];
            for(int i=0; i<max_nodes; ++i) {
                if(p1_ptr[i*NODE_INFO_DIM + COL_PARENT_IDX] != -1 && get_root_branch_type_device(p1_ptr, i, max_nodes) == branch_type_to_try) p1_candidates[p1_cand_count++] = i;
                if(p2_ptr[i*NODE_INFO_DIM + COL_PARENT_IDX] != -1 && get_root_branch_type_device(p2_ptr, i, max_nodes) == branch_type_to_try) p2_candidates[p2_cand_count++] = i;
            }
        }

        if (p1_cand_count == 0 || p2_cand_count == 0) continue;
        
        int p1_idx = p1_candidates[(int)(hiprand_uniform(&state) * p1_cand_count)];
        int p2_idx = p2_candidates[(int)(hiprand_uniform(&state) * p2_cand_count)];

        // --- 3. 서브트리 정보 수집 ---
        int s1_count = find_subtree_nodes_device(p1_ptr, p1_idx, my_queue1, my_results1, max_nodes);
        int s2_count = find_subtree_nodes_device(p2_ptr, p2_idx, my_queue2, my_results2, max_nodes);

        int p1_total_nodes = get_active_node_count(p1_ptr, max_nodes);
        int p2_total_nodes = get_active_node_count(p2_ptr, max_nodes);
        
        // --- 4. 제약 조건 검증 ---
        int p1_parent_idx = (int)p1_ptr[p1_idx*NODE_INFO_DIM + COL_PARENT_IDX];
        if(p1_parent_idx < 0 || p1_parent_idx >= max_nodes) continue;
        float p1_ins_depth = p1_ptr[p1_parent_idx*NODE_INFO_DIM + COL_DEPTH] + 1;
        if(p1_ins_depth + get_max_relative_depth(p2_ptr, my_results2, s2_count, max_nodes) > max_depth) continue;
        if(p1_total_nodes - s1_count + s2_count > max_nodes) continue;

        int p2_parent_idx = (int)p2_ptr[p2_idx*NODE_INFO_DIM + COL_PARENT_IDX];
        if(p2_parent_idx < 0 || p2_parent_idx >= max_nodes) continue;
        float p2_ins_depth = p2_ptr[p2_parent_idx*NODE_INFO_DIM + COL_DEPTH] + 1;
        if(p2_ins_depth + get_max_relative_depth(p1_ptr, my_results1, s1_count, max_nodes) > max_depth) continue;
        if(p2_total_nodes - s2_count + s1_count > max_nodes) continue;

        // --- 5. 이식 수행 ---
        bool transplant1_success = transplant_one_way_device(c1_ptr, p1_ptr, p2_ptr,
            p1_idx, p2_idx,
            my_results1, s1_count,
            my_results2, s2_count,
            my_old_to_new_map, my_queue1, max_nodes);

        bool transplant2_success = transplant_one_way_device(c2_ptr, p2_ptr, p1_ptr,
            p2_idx, p1_idx,
            my_results2, s2_count,
            my_results1, s1_count,
            my_old_to_new_map, my_queue2, max_nodes);
        
        success = transplant1_success && transplant2_success;
    }

    if (!success) {
        for(int i=0; i < max_nodes * NODE_INFO_DIM; ++i) c1_ptr[i] = p1_ptr[i];
        for(int i=0; i < max_nodes * NODE_INFO_DIM; ++i) c2_ptr[i] = p2_ptr[i];
    }
}

// ==============================================================================
//                       C++ 래퍼 함수 (커널 런처)
// ==============================================================================
void get_contextual_mask_cuda(const torch::Tensor& trees, torch::Tensor& output_mask, int node_type, int branch_type) {
    const int batch_size = trees.size(0);
    const int max_nodes = trees.size(1);
    const int total_nodes = batch_size * max_nodes;
    if (total_nodes == 0) return;

    const int threads = 256;
    const int blocks = (total_nodes + threads - 1) / threads;
    
    get_contextual_mask_kernel<<<blocks, threads>>>(
        trees.data_ptr<float>(),
        output_mask.data_ptr<bool>(),
        batch_size, max_nodes, node_type, branch_type);
    hipDeviceSynchronize();
}

void swap_node_params_cuda(torch::Tensor& c1, torch::Tensor& c2, const torch::Tensor& p1_mask, const torch::Tensor& p2_mask) {
    const int batch_size = c1.size(0);
    if (batch_size == 0) return;
    const int max_nodes = c1.size(1);

    swap_node_params_kernel<<<batch_size, 1>>>(
        c1.data_ptr<float>(),
        c2.data_ptr<float>(),
        p1_mask.data_ptr<bool>(),
        p2_mask.data_ptr<bool>(),
        batch_size, max_nodes);
    hipDeviceSynchronize();
}

void copy_branches_batch_cuda(
    torch::Tensor& child_batch,
    const torch::Tensor& p1_batch,
    const torch::Tensor& p2_batch,
    const torch::Tensor& donor_map,
    torch::Tensor& bfs_queue_buffer,
    torch::Tensor& result_indices_buffer,
    torch::Tensor& old_to_new_map_buffer
)
{
    const int batch_size = child_batch.size(0);
    if (batch_size == 0) return;
    const int max_nodes = child_batch.size(1);

    copy_branches_kernel<<<batch_size, 1>>>(
        child_batch.data_ptr<float>(),
        p1_batch.data_ptr<float>(),
        p2_batch.data_ptr<float>(),
        donor_map.data_ptr<int>(),
        bfs_queue_buffer.data_ptr<int>(),
        result_indices_buffer.data_ptr<int>(),
        old_to_new_map_buffer.data_ptr<int>(),
        batch_size,
        max_nodes
    );
    hipDeviceSynchronize();
}

// [수정된] SubtreeCrossover 래퍼 함수
void subtree_crossover_batch_cuda(
    torch::Tensor& child1_out,
    torch::Tensor& child2_out,
    const torch::Tensor& p1_batch,
    const torch::Tensor& p2_batch,
    int mode,
    int max_depth,
    int max_nodes,
    int max_retries,
    const torch::Tensor& branch_perm,
    torch::Tensor& bfs_queue_buffer,
    torch::Tensor& result_indices_buffer,
    torch::Tensor& old_to_new_map_buffer,
    torch::Tensor& p1_candidates_buffer,
    torch::Tensor& p2_candidates_buffer)
{
    const int batch_size = p1_batch.size(0);
    if (batch_size == 0) return;

    // [중요] 버퍼 크기 검증: 큐/결과는 2*max_nodes, 그 외는 >= max_nodes
    TORCH_CHECK(bfs_queue_buffer.dim() >= 2 && bfs_queue_buffer.size(1) >= 2*max_nodes,
                "bfs_queue_buffer must have second dim >= 2*max_nodes");
    TORCH_CHECK(result_indices_buffer.dim() >= 2 && result_indices_buffer.size(1) >= 2*max_nodes,
                "result_indices_buffer must have second dim >= 2*max_nodes");
    TORCH_CHECK(old_to_new_map_buffer.dim() >= 2 && old_to_new_map_buffer.size(1) >= max_nodes,
                "old_to_new_map_buffer must have second dim >= max_nodes");
    TORCH_CHECK(p1_candidates_buffer.dim() >= 2 && p1_candidates_buffer.size(1) >= max_nodes,
                "p1_candidates_buffer must have second dim >= max_nodes");
    TORCH_CHECK(p2_candidates_buffer.dim() >= 2 && p2_candidates_buffer.size(1) >= max_nodes,
                "p2_candidates_buffer must have second dim >= max_nodes");

    subtree_crossover_kernel<<<batch_size, 1>>>(
        child1_out.data_ptr<float>(),
        child2_out.data_ptr<float>(),
        p1_batch.data_ptr<float>(),
        p2_batch.data_ptr<float>(),
        mode, max_depth, max_nodes, max_retries,
        branch_perm.data_ptr<int>(),
        bfs_queue_buffer.data_ptr<int>(),
        result_indices_buffer.data_ptr<int>(),
        old_to_new_map_buffer.data_ptr<int>(),
        p1_candidates_buffer.data_ptr<int>(),
        p2_candidates_buffer.data_ptr<int>(),
        batch_size
    );
    hipDeviceSynchronize();
}